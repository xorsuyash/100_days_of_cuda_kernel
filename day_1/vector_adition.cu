#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addVectors(float *a, float *b, float *c, int n)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if (index<n){
        c[index]=a[index]+b[index];
    }
}

int main(void){

    int n=10;
    size_t bytes=n*sizeof(float);

    float *h_a,*h_b,*h_c;

    float *d_a,*d_b,*d_c;

    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    for (int i=0; i<n; i++){
        h_a[i] = i;
        h_b[i] = i*2;
    }

    hipMalloc(&d_a,bytes);
    hipMalloc(&d_b,bytes);
    hipMalloc(&d_c,bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize=256;
    int gridSize = (n+blockSize-1)/blockSize;


    addVectors<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    
    printf("Vector addition results:\n");
    for (int i = 0; i < n; i++) {
        printf("%0.1f + %0.1f = %0.1f\n", h_a[i], h_b[i], h_c[i]);
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    
    return 0;



}